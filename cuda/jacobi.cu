#include "hip/hip_runtime.h"
#include <lcutil.h>
#include <timestamp.h>
#include <stdio.h>
#include <math.h>
#include "util.h"

#define BLOCK_SIZE 256
#define XRIGHT 1
#define XLEFT -1
#define YTOP 1
#define YBOTTOM -1

#define ROUND_UP(XX, YY) ((double) (XX + YY - 1) / YY);


struct Neighbors {
	int north;
	int south;
	int west;
	int east;
	int center;
};

__device__ Neighbors constructNeighbors(int i, int columns);
__device__ __attribute__((always_inline)) inline void calculateOneElement(const int y, const int x, 
	const struct Neighbors *neighbors, const double *sharedArray, double *array, 
	const struct JacobiParams jacobiParams, double *errorArray, const int inputColumns, const int inputRows);

__global__ void kjacobi(double *array, double *newArray, int inputRows, int inputColumns, const JacobiParams jacobiParams, const int sharedMemorySize, double *errorArray) {
	const unsigned int column = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int i = row * inputColumns + column;  

	extern __shared__ double sharedArray[];
	const unsigned int sharedDimX = blockDim.x + 2;

	const unsigned int columnShared = threadIdx.x + 1;
	const unsigned int rowShared = threadIdx.y + 1;
	const unsigned int iShared = rowShared * sharedDimX + columnShared; 

	struct Neighbors arrayNeighbors = constructNeighbors(i, inputColumns);
	struct Neighbors sharedArrayNeighbors = constructNeighbors(iShared, sharedDimX);
	
	if (column < inputColumns && row < inputRows) {
		sharedArray[iShared] = array[i];

		// Compute halo points. Set 0 when there isn't neighbor block, or when we're at the end of array within the last block 
		if (!threadIdx.y) { // First row of block
			sharedArray[sharedArrayNeighbors.north] = (blockIdx.y > 0 ? array[arrayNeighbors.north] : 0);
		}
		if (threadIdx.y == blockDim.y - 1 || row == inputRows - 1) { // Last row of block OR last row of array 
			sharedArray[sharedArrayNeighbors.south] = (blockIdx.y < gridDim.y - 1 && row < inputRows - 1 ? array[arrayNeighbors.south] : 0);
		}
		if (!threadIdx.x) { // First column of block
			sharedArray[sharedArrayNeighbors.west] = (blockIdx.x > 0 ? array[arrayNeighbors.west] : 0);
		}
		if (threadIdx.x == blockDim.x - 1 || column == inputColumns - 1) { // Last column of block OR last column of array
			sharedArray[sharedArrayNeighbors.east] = (blockIdx.x < gridDim.x - 1 && column < inputColumns - 1 ? array[arrayNeighbors.east] : 0);
		}
	}

	__syncthreads();

	if (column < inputColumns && row < inputRows) {
		calculateOneElement(row, column, &sharedArrayNeighbors, sharedArray, &newArray[i], jacobiParams, &errorArray[i], inputColumns, inputRows);
	}

	__syncthreads();
}

__global__ void ksum(double *array, double *newArray) {
	const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int iShared = threadIdx.x;
	extern __shared__ double sharedArray[];
	sharedArray[iShared] = array[i];
	//printf("error[%d] %.15f\n",  i, array[i]);
	__syncthreads();

	int threads = ROUND_UP(blockDim.x, 2);
	while (threads > 0) {
		if (threadIdx.x < threads) {
			int first = iShared * 2;
			sharedArray[iShared] = sharedArray[first] + sharedArray[first + 1];
			//printf("Block %d thread %d sum %.15f + %.15f\n", blockIdx.x, threadIdx.x, sharedArray[first], sharedArray[first+1]);
		}		
		__syncthreads();
		threads /= 2;
	}
	
	if (!threadIdx.x) {
		newArray[blockIdx.x] = sharedArray[iShared];
		//printf("Block %d %.15f\n", blockIdx.x, newArray[i]);
	}
}

extern "C" float jacobiGPU(double *array, int elements, int inputRows, int inputColumns, JacobiParams jacobiParams) {
	double *arrayDevice;
	double finalError;
	hipError_t err;
	int arrayBytes = elements * sizeof(double);

	int rowsOfBlocks, columnsOfBlocks, rowsOfBlockThreads, columnsOfBlockThreads;
	divide2D(BLOCK_SIZE, &rowsOfBlockThreads, &columnsOfBlockThreads);
	rowsOfBlocks = ceil((float) inputRows / rowsOfBlockThreads);
	columnsOfBlocks = ceil((float) inputColumns / columnsOfBlockThreads);

	int sharedMemorySize = (rowsOfBlockThreads + 2) * (columnsOfBlockThreads + 2);
	
	dim3 dimBl(columnsOfBlockThreads, rowsOfBlockThreads);
	dim3 dimGr(columnsOfBlocks, rowsOfBlocks);

	printf("Elements %d\nGrid %d X %d\nBlock threads %d X %d\n", elements, rowsOfBlocks, columnsOfBlocks, rowsOfBlockThreads, columnsOfBlockThreads);

	err = hipMalloc((void **)&arrayDevice, arrayBytes);
	if (err != hipSuccess){
		fprintf(stderr, "GPUassert: %s\n",err);
		return err;
	}

	// Copy data to device memory
	err = hipMemcpy(arrayDevice, array, arrayBytes, hipMemcpyHostToDevice);
	if (err != hipSuccess){
		fprintf(stderr, "GPUassert: %s\n",err);
		return err;
	}

	double *newArrayDevice;

	err = hipMalloc((void **)&newArrayDevice, arrayBytes);
	if (err != hipSuccess){
		fprintf(stderr, "GPUassert: %s\n",err);
		return err;
	}

	// Copy data to device memory
	err = hipMemcpy(newArrayDevice, array, arrayBytes, hipMemcpyHostToDevice);
	if (err != hipSuccess){
		fprintf(stderr, "GPUassert: %s\n",err);
		return err;
	}

	double *errorArray = (double *) malloc(arrayBytes);
	double *errorArrayDevice;
	err = hipMalloc((void **)&errorArrayDevice, arrayBytes);
	if (err != hipSuccess){
		fprintf(stderr, "GPUassert: %s\n",err);
		return err;
	}

	double *newErrorArrayDevice;
	err = hipMalloc((void **)&newErrorArrayDevice, arrayBytes);
	if (err != hipSuccess){
		fprintf(stderr, "GPUassert: %s\n",err);
		return err;
	}

	double *srcArray = arrayDevice;
	double *dstArray = newArrayDevice;

	int iterations = jacobiParams.maxIterations;
	timestamp t_start;
	t_start = getTimestamp();

	while(iterations--) {
		kjacobi<<<dimGr, dimBl, sharedMemorySize * sizeof(double)>>>(srcArray, dstArray, inputRows, inputColumns, jacobiParams, sharedMemorySize, errorArrayDevice);
		err = hipGetLastError();
		if (err != hipSuccess){
			fprintf(stderr, "GPUassert: %s\n",err);
			return err;
		}
	
		err = hipDeviceSynchronize();
		if (err != hipSuccess){
			fprintf(stderr, "GPUassert: %s\n",err);
			return err;
		}

		err = hipMemcpy(errorArray, errorArrayDevice, arrayBytes, hipMemcpyDeviceToHost);
		if (err != hipSuccess){
			fprintf(stderr, "GPUassert: %s\n",err);
			return err;
		}
		
		for (int i = 0 ; i < elements ; i++) {
			;//printf("CPU error[%d]: %.15f\n", i, errorArray[i]);
		}

		if (jacobiParams.checkConvergence) {
			double *srcErrorArray = errorArrayDevice;
			double *dstErrorArray = newErrorArrayDevice;
			int errorElements = elements;

			while (errorElements > 1) {
				int errorBlocks = ceil((double) errorElements / BLOCK_SIZE);
				//printf("blocks %d elements %d \n", errorBlocks, errorElements);
				
				dim3 dimBlSum(BLOCK_SIZE);
				dim3 dimGrSum(errorBlocks);
				ksum<<<dimGrSum, dimBlSum, BLOCK_SIZE * sizeof(double)>>>(srcErrorArray, dstErrorArray);
				
				err =hipGetLastError();
				if (err != hipSuccess){
						fprintf(stderr, "GPUassert: %s\n",err);
						return err;
				}
				err = hipDeviceSynchronize();
				if (err != hipSuccess){
					fprintf(stderr, "GPUassert: %s\n",err);
					return err;
				}

				errorElements = errorBlocks;
				//return 0; ///////

				if (errorElements > 1) {
					double *tmp = srcErrorArray;
					srcErrorArray = dstErrorArray;
					dstErrorArray = tmp;
				}
			}

			err = hipMemcpy(errorArray, dstErrorArray, sizeof(double), hipMemcpyDeviceToHost);
			if (err != hipSuccess){
				fprintf(stderr, "GPUassert: %s\n",err);
				return err;
			}

			finalError = sqrt(errorArray[0])/(inputColumns * inputRows);
			if (finalError <= jacobiParams.tol) {
				break;
			}
		}

		if (iterations) {
			double *tmp = srcArray;
			srcArray = dstArray;
			dstArray = tmp;
		}
	}

	float msecs = getElapsedtime(t_start);
	

	err = hipMemcpy(array, dstArray, arrayBytes, hipMemcpyDeviceToHost);
	if (err != hipSuccess){
		fprintf(stderr, "GPUassert: %s\n",err);
		return err;
	}

	hipFree(arrayDevice);

	printf("Error is %g\n", finalError);

	return msecs;
}

__device__ __attribute__((always_inline)) inline void calculateOneElement(const int y, const int x, 
	const struct Neighbors *neighbors, const double *sharedArray, double *array, 
	const struct JacobiParams jacobiParams, double *errorArray, const int inputColumns, const int inputRows) {

	// Jacobi constants
	const double deltaX = (double) (XRIGHT - XLEFT) / (inputColumns - 1);
	const double deltaY = (double) (YTOP - YBOTTOM) / (inputRows - 1);
	const double cx = 1.0/(deltaX*deltaX);
	const double cy = 1.0/(deltaY*deltaY);
	const double cc = -2.0*cx-2.0*cy-jacobiParams.alpha;

	double fY = YBOTTOM + (y)*deltaY;
	double fYSquare = fY*fY;
	double fX = XLEFT + (x)*deltaX;
	double fXSquare = fX*fX;

	double f = -jacobiParams.alpha*(1.0-fXSquare)*(1.0-fYSquare) - 2.0*(1.0-fXSquare) - 2.0*(1.0-fYSquare);
	double curVal = sharedArray[neighbors->center];
	double updateVal = ((sharedArray[neighbors->west] + sharedArray[neighbors->east])*cx +
	(sharedArray[neighbors->north] + sharedArray[neighbors->south])*cy + curVal*cc - f
	)/cc;
	
	*array = curVal - jacobiParams.relax*updateVal;
	*errorArray = updateVal*updateVal;
	//printf("%f\n", *errorArray);
}

__device__ Neighbors constructNeighbors(int i, int columns) {
	struct Neighbors neighbors;
	neighbors.center = i;
	neighbors.north = i - columns;
	neighbors.south = i + columns;
	neighbors.west = i - 1;
	neighbors.east = i + 1;

	return neighbors;
}